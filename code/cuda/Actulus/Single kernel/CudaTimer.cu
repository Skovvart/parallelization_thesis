#include <stdio.h>
#include <hip/hip_runtime.h>

hipEvent_t start, stop;
float timertime;

void timerstart() {
	timertime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

float timerstop() {
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timertime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return timertime;
}